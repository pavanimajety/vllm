#include "hip/hip_runtime.h"
#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

#include <iostream>

#include <cutlass/array.h>
#include <cutlass/numeric_types.h>

#include "../../../moe/permute_unpermute_kernels/dispatch.h"


constexpr uint64_t THREADS_PER_EXPERT = 512;

__global__ void compute_problem_sizes(const int* __restrict__ topk_ids,
                                      int32_t* problem_sizes1,
                                      int32_t* problem_sizes2,
                                      int32_t* atomic_buffer,
                                      const int topk_length, const int n,
                                      const int k) {
  int expert_id = blockIdx.x;

  int occurrences = 0;
  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    occurrences += (topk_ids[i] == expert_id);
  }
  atomicAdd(&atomic_buffer[expert_id], occurrences);
  __syncthreads();

  if (threadIdx.x == 0) {
    int final_occurrences = atomic_buffer[expert_id];
    problem_sizes1[expert_id * 3] = final_occurrences;
    problem_sizes1[expert_id * 3 + 1] = 2 * n;
    problem_sizes1[expert_id * 3 + 2] = k;
    problem_sizes2[expert_id * 3] = final_occurrences;
    problem_sizes2[expert_id * 3 + 1] = k;
    problem_sizes2[expert_id * 3 + 2] = n;
  }
}

__global__ void compute_expert_offsets(
    const int32_t* __restrict__ problem_sizes1, int32_t* expert_offsets,
    int32_t* atomic_buffer, const int num_experts) {
  int32_t tot_offset = 0;
  expert_offsets[0] = 0;
  for (int i = 0; i < num_experts; ++i) {
    atomic_buffer[i] = tot_offset;
    tot_offset += problem_sizes1[i * 3];
    expert_offsets[i + 1] = tot_offset;
  }
}

__global__ void compute_expert_blockscale_offsets(
    const int32_t* __restrict__ problem_sizes1, int32_t* expert_offsets, int32_t* blockscale_offsets, 
    int32_t* atomic_buffer, const int num_experts) {
  int32_t tot_offset = 0;
  int32_t tot_offset_round = 0;
  expert_offsets[0] = 0;
  blockscale_offsets[0] = 0;
  for (int i = 0; i < num_experts; ++i) {
    atomic_buffer[i] = tot_offset;
    tot_offset += problem_sizes1[i * 3];
    expert_offsets[i + 1] = tot_offset;
    tot_offset_round += (problem_sizes1[i * 3] + (128 - 1)) / 128 * 128;
    blockscale_offsets[i + 1] = tot_offset_round;
  }
}

__global__ void compute_arg_sorts(const int* __restrict__ topk_ids,
                                  const int32_t* __restrict__ expert_offsets,
                                  int32_t* input_permutation,
                                  int32_t* output_permutation,
                                  int32_t* atomic_buffer, const int topk_length,
                                  const int topk) {
  int const blk_expert_id = blockIdx.x;
  int const num_experts = gridDim.x;
  int32_t const num_tokens = expert_offsets[num_experts];

  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    int const expert_id = topk_ids[i];
    if (expert_id == -1 && blockIdx.x == 0) {
      // output_permutation is used to re-order the moe outputs. It is
      // used as c2 = c2[c_map], where c2 is a torch.tensor that is the
      // output of the cutlass kernels and c_map is the output_permutation.
      // c2 is initialized to zeros, therefore by setting the output_permutation
      // to num_tokens, we are guaranteed to fill the moe outputs to zero
      // for "invalid" topk_ids.
      output_permutation[i] = num_tokens;
    } else if (expert_id == blk_expert_id) {
      int start = atomicAdd(&atomic_buffer[expert_id], 1);
      input_permutation[start] = i / topk;
      output_permutation[i] = start;
    }
  }
}

void get_cutlass_moe_mm_data_caller(
    const torch::Tensor& topk_ids, torch::Tensor& expert_offsets,
    torch::Tensor& problem_sizes1, torch::Tensor& problem_sizes2,
    torch::Tensor& input_permutation, torch::Tensor& output_permutation,
    const int64_t num_experts, const int64_t n, const int64_t k, const std::optional<torch::Tensor>& blockscale_offsets) {
  auto stream = at::cuda::getCurrentCUDAStream(topk_ids.device().index());
  auto options_int32 =
      torch::TensorOptions().dtype(torch::kInt32).device(topk_ids.device());
  torch::Tensor atomic_buffer = torch::zeros(num_experts, options_int32);

  int num_threads = min(THREADS_PER_EXPERT, topk_ids.numel());
  compute_problem_sizes<<<num_experts, num_threads, 0, stream>>>(
      static_cast<const int32_t*>(topk_ids.data_ptr()),
      static_cast<int32_t*>(problem_sizes1.data_ptr()),
      static_cast<int32_t*>(problem_sizes2.data_ptr()),
      static_cast<int32_t*>(atomic_buffer.data_ptr()), topk_ids.numel(), n, k);
  if (blockscale_offsets.has_value()) {
    compute_expert_blockscale_offsets<<<1, 1, 0, stream>>>(
        static_cast<const int32_t*>(problem_sizes1.data_ptr()),
        static_cast<int32_t*>(expert_offsets.data_ptr()),
        static_cast<int32_t*>(blockscale_offsets.value().data_ptr()),
        static_cast<int32_t*>(atomic_buffer.data_ptr()), num_experts);
  }
  else {
  compute_expert_offsets<<<1, 1, 0, stream>>>(
      static_cast<const int32_t*>(problem_sizes1.data_ptr()),
      static_cast<int32_t*>(expert_offsets.data_ptr()),
      static_cast<int32_t*>(atomic_buffer.data_ptr()), num_experts);
  }
  compute_arg_sorts<<<num_experts, num_threads, 0, stream>>>(
      static_cast<const int32_t*>(topk_ids.data_ptr()),
      static_cast<const int32_t*>(expert_offsets.data_ptr()),
      static_cast<int32_t*>(input_permutation.data_ptr()),
      static_cast<int32_t*>(output_permutation.data_ptr()),
      static_cast<int32_t*>(atomic_buffer.data_ptr()), topk_ids.numel(),
      topk_ids.size(1));
}


template <typename T>
__global__ void expandInputRowsKernel(
    const T* input, const int32_t* dst2src_map, T* output,
    int64_t num_src_rows, int64_t num_dst_rows, int64_t num_cols) {
  int64_t dest_row_idx = blockIdx.x;
  int64_t const source_row_idx = dst2src_map[dest_row_idx];

  if (blockIdx.x < num_dst_rows) {
    // Load 128-bits per thread
    constexpr int64_t ELEM_PER_THREAD = 128 / sizeof(T) / 8;
    using DataElem = cutlass::Array<T, ELEM_PER_THREAD>;

    // Duplicate and permute rows
    auto const* source_row_ptr =
        reinterpret_cast<DataElem const*>(input + source_row_idx * num_cols);
    auto* dest_row_ptr =
        reinterpret_cast<DataElem*>(output + dest_row_idx * num_cols);

    int64_t const start_offset = threadIdx.x;
    int64_t const stride = blockDim.x;
    int64_t const num_elems_in_col = num_cols / ELEM_PER_THREAD;

    for (int elem_index = start_offset; elem_index < num_elems_in_col;
         elem_index += stride) {
      dest_row_ptr[elem_index] = source_row_ptr[elem_index];
    }
  }
}

void moe_permute_caller(
    const torch::Tensor& input_tensor,
    const torch::Tensor& dst2src_map,
    torch::Tensor& output_tensor) {
  TORCH_CHECK(input_tensor.scalar_type() == output_tensor.scalar_type(),
              "Input and output tensors must have the same data type");

  auto stream = at::cuda::getCurrentCUDAStream().stream();
  int64_t const blocks = output_tensor.size(0);
  int64_t const threads = 256;
  int64_t const num_dest_rows = output_tensor.size(0);
  int64_t const num_src_rows = input_tensor.size(0);
  int64_t const num_cols = input_tensor.size(1);

  MOE_DISPATCH(input_tensor.scalar_type(), [&] {
        expandInputRowsKernel<scalar_t><<<blocks, threads, 0, stream>>>(
            reinterpret_cast<scalar_t*>(input_tensor.data_ptr()),
            dst2src_map.data_ptr<int32_t>(),
            reinterpret_cast<scalar_t*>(output_tensor.data_ptr()),
            num_src_rows,
            num_dest_rows,
            num_cols);
      });
}